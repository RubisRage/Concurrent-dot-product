#include "hip/hip_runtime.h"
#define CUDA /* Tell the preprocessor this is a CUDA/C++ file */

#include "matrix-utils.h"
#include <stdio.h>
#include <stdlib.h>

#define MAX_THREADS_PER_BLOCK 1024

__global__ void __dot_product(double* a, double* b, double* c,
                              size_t n, size_t p, size_t m){
    size_t index = threadIdx.x + blockIdx.x*blockDim.x;

    if(index >= n*m) return;

    double v = 0.0;
    size_t i = index/m, j = index%m;

    for(size_t offset = 0; offset<p; offset++){
        v += *(a + i*p + offset) * *(b + m*offset + j);
    }

    c[index] = v;
}

extern "C" MATRIX dot_product(double* a, double* b, size_t n, size_t p, size_t m){
    double *da, *db, *dc;
    double* c = (double*)malloc(sizeof(double)*n*m);
    if(c==NULL) {
        perror("CUDA: while allocating memory");
        exit(EXIT_FAILURE);
    }

    int blocks = (n*m)/MAX_THREADS_PER_BLOCK + 1;

    hipMalloc((void**)&da, n*p*sizeof(double));
    hipMalloc((void**)&db, p*m*sizeof(double));
    hipMalloc((void**)&dc, n*m*sizeof(double));

    hipMemcpy(da, a, n*p*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(db, b, p*m*sizeof(double), hipMemcpyHostToDevice);

    __dot_product<<<blocks,MAX_THREADS_PER_BLOCK>>>(da, db, dc, n, p, m);
    hipError_t err;
    if((err=hipDeviceSynchronize())!=hipSuccess){
    }

    hipMemcpy(c, dc, n*m*sizeof(double), hipMemcpyDeviceToHost);
    MATRIX mat = {.data=c, .n=n, .m=m};

    hipFree(da); hipFree(db); hipFree(dc);

    return mat;
}

