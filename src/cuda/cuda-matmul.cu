#include "hip/hip_runtime.h"
#define CUDA /* Tell the preprocessor this is a CUDA/C++ file */

#include "matrix-utils.h"
#include <stdio.h>
#include <stdlib.h>

#define MAX_THREADS_PER_BLOCK 1024

/*
 * CUDA Kernel for computing the matrix multiplication.
 *
 * @param a Double pointer referring to the first matrix.
 * @param b Double pointer referring to the second matrix.
 * @param c Double pointer referring to device allocated memory
 * for the result matrix.
 * @param n Number of rows of a.
 * @param p Number of columns of a and rows of b.
 * @param m Number of columns of b.
 * */
__global__ void __dot_product(double* a, double* b, double* c,
                              size_t n, size_t p, size_t m){
    size_t index = threadIdx.x + blockIdx.x*blockDim.x;

    if(index >= n*m) return;

    double v = 0.0;
    size_t i = index/m, j = index%m;

    for(size_t offset = 0; offset<p; offset++){
        v += *(a + i*p + offset) * *(b + m*offset + j);
    }

    c[index] = v;
}

/*
 * Computes the matrix multiplication between the matrices pointed by a and b.
 *
 * @param a Double pointer referring to the first matrix.
 * @param b Double pointer referring to the second matrix.
 * @param n Number of rows of a.
 * @param p Number of columns of a and rows of b.
 * @param m Number of columns of b.
 *
 * @return MATRIX structure containing the matrix multiplication.
 * */
extern "C" MATRIX dot_product(double* a, double* b, size_t n, size_t p, size_t m){
    double *da, *db, *dc;
    double* c = (double*)malloc(sizeof(double)*n*m);
    if(c==NULL) {
        perror("CUDA: while allocating memory");
        exit(EXIT_FAILURE);
    }

    int blocks = (n*m)/MAX_THREADS_PER_BLOCK + 1;

    hipMalloc((void**)&da, n*p*sizeof(double));
    hipMalloc((void**)&db, p*m*sizeof(double));
    hipMalloc((void**)&dc, n*m*sizeof(double));

    hipMemcpy(da, a, n*p*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(db, b, p*m*sizeof(double), hipMemcpyHostToDevice);

    __dot_product<<<blocks,MAX_THREADS_PER_BLOCK>>>(da, db, dc, n, p, m);

    hipMemcpy(c, dc, n*m*sizeof(double), hipMemcpyDeviceToHost);
    MATRIX mat = {.data=c, .n=n, .m=m};

    hipFree(da); hipFree(db); hipFree(dc);

    return mat;
}

