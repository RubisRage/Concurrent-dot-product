#include "hip/hip_runtime.h"
#define CUDA /* Tell the preprocessor this is a CUDA/C++ file */

#include "matrix-utils.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define THREADS_PER_DIM 16 /* For 2 dimensions 32*32=1024, max # of threads/block */

/*
 * CUDA Kernel for computing the matrix multiplication.
 *
 * @param a Double pointer referring to the first matrix.
 * @param b Double pointer referring to the second matrix.
 * @param c Double pointer referring to device allocated memory
 * for the result matrix.
 * @param n Number of rows of a.
 * @param p Number of columns of a and rows of b.
 * @param m Number of columns of b.
 * */
__global__ void __dot_product(double* a, double* b, double* c,
                              size_t n, size_t p, size_t m){

    size_t i = blockIdx.y*blockDim.y + threadIdx.y;
    size_t j = blockIdx.x*blockDim.x + threadIdx.x;

    if(i >= n || j >= m){
        return;
    }

    double v = 0.0;
    for(size_t offset = 0; offset < p; offset++){
        v += a[i*p + offset] * b[m*offset + j];
    }

    c[i*m + j] = v;
}

/*
 * Computes the matrix multiplication between the matrices pointed by a and b.
 *
 * @param a Double pointer referring to the first matrix.
 * @param b Double pointer referring to the second matrix.
 * @param n Number of rows of a.
 * @param p Number of columns of a and rows of b.
 * @param m Number of columns of b.
 *
 * @return MATRIX structure containing the matrix multiplication.
 * */
extern "C" MATRIX dot_product(double* a, double* b, size_t n, size_t p, size_t m){
    double *da, *db, *dc;
    double* c = (double*)malloc(sizeof(double)*n*m);
    if(c==NULL) {
        perror("CUDA: while allocating memory");
        exit(EXIT_FAILURE);
    }

    dim3 block_dims(THREADS_PER_DIM, THREADS_PER_DIM, 1);   /* Blocks of 32*32        */
    dim3 grid_dims((int)ceil(n/(double)THREADS_PER_DIM),    /* Blocks per grid row    */
                   (int)ceil(m/(double)THREADS_PER_DIM),1); /* Blocks per grid column */


    hipMalloc((void**)&da, n*p*sizeof(double));
    hipMalloc((void**)&db, p*m*sizeof(double));
    hipMalloc((void**)&dc, n*m*sizeof(double));

    hipMemcpy(da, a, n*p*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(db, b, p*m*sizeof(double), hipMemcpyHostToDevice);

    __dot_product<<<grid_dims,block_dims>>>(da, db, dc, n, p, m);

    hipMemcpy(c, dc, n*m*sizeof(double), hipMemcpyDeviceToHost);
    MATRIX mat = {.data=c, .n=n, .m=m};

    hipFree(da); hipFree(db); hipFree(dc);

    return mat;
}

